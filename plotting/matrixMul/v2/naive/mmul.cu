// This program computes matrix multiplication
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <cstdlib>

// Matrix Multiplication kernel
// Optimizations:
__global__ void matrixMul(int *a, int *b, int *c, int N){
    // Calculate the row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if((row < N) && (col < N)){
        // Each thread computes one element
        for(int i = 0; i < N; i++){
            c[row * N + col] += a[row * N + i] * b[i * N + col];
        }
    }
}

// Initialize a matrix with random numbers
void init_matrix(int *m, int N){
    for(int i = 0; i < N * N; i++){
        m[i] = rand() % 100;
    }
}

// Verify result (only needs to be run once to ensure functional
// correctness)
void verify_result(int *a, int *b, int *c, int N){
    int tmp = 0;
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            for(int k = 0; k < N; k++){
                tmp += a[i * N + k] * b[k * N + j];
            }
            assert(c[i * N + j] == tmp);
        }
    }
}

int main(){
    // Problem size
    int N = 1 << 14;
    size_t bytes = N * N * sizeof(int);

    // Allocate host memory (make sure C is zeroed)
    int *h_a = (int*)malloc(bytes);
    int *h_b = (int*)malloc(bytes);
    int *h_c = (int*)calloc(N * N, sizeof(int));
    
    // Allocate device memory
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Initialized host-side matrices
    init_matrix(h_a, N * N);
    init_matrix(h_b, N * N);

    // Copy the matrices over
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);

    // Set up the CTA and Grid Dimensions
    int threads = 16;
    int blocks = (N + threads -1) / threads;
    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    // Call our kernel
    matrixMul<<<BLOCKS, THREADS>>>(d_a, d_b, d_c, N);
    
    // Copy data back to the host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    return 0;
}
